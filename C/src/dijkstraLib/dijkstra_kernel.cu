#include "hip/hip_runtime.h"
//
//
//  Description:
//      Implementation of Dijkstra's Single-Source Shortest Path (SSSP) algorithm on the GPU.
//      The basis of this implementation is the paper:
//
//          "Accelerating large graph algorithms on the GPU using CUDA" by
//          Parwan Harish and P.J. Narayanan
//
//
//  Author:
//      Dan Ginsburg
//
//  Children's Hospital Boston
//  GPL v2
//
// includes, project
#include <cutil_inline.h>
#include <stdio.h>
#include <float.h>
#include <multithreading.h>

#include "dijkstra_kernel.h"

///
//  Types
//

// This structure is used in the multi-GPU implementation of the algorithm.
// This structure defines the workload for each GPU.  The code chunks up
// the work on a per-GPU basis.
typedef struct
{
    // GPU number to run algorithm on
    int device;

    // Pointer to graph data
    GraphData *graph;

    // Source vertex indices to process
    int *sourceVertices;

    // End vertex indices to process
    int *endVertices;

    // Results of processing
    float *outResultCosts;

    // Number of results
    int numResults;

} GPUPlan;

///////////////////////////////////////////////////////////////////////////////
//
//  CUDA Compute Kernels
//
//

///
/// This is part 1 of the Kernel from Algorithm 4 in the paper
///
__global__  void CUDA_SSSP_KERNEL1( int *vertexArray, int *edgeArray, float *weightArray,
                                    unsigned char *maskArray, float *costArray, float *updatingCostArray,
                                    int vertexCount, int edgeCount )
{
    // access thread id
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( maskArray[tid] != 0 )
    {
        maskArray[tid] = 0;

        int edgeStart = vertexArray[tid];
        int edgeEnd;
        if (tid + 1 < (vertexCount))
        {
            edgeEnd = vertexArray[tid + 1];
        }
        else
        {
            edgeEnd = edgeCount;
        }

        for(int edge = edgeStart; edge < edgeEnd; edge++)
        {
            int nid = edgeArray[edge];

            // One note here: whereas the paper specified weightArray[nid], I
            //  found that the correct thing to do was weightArray[edge].  I think
            //  this was a typo in the paper.  Either that, or I misunderstood
            //  the data structure.
            if (updatingCostArray[nid] > (costArray[tid] + weightArray[edge]))
            {
                updatingCostArray[nid] = (costArray[tid] + weightArray[edge]);
            }
        }
    }
}

///
/// This is part 2 of the Kernel from Algorithm 5 in the paper.  The only modification
/// is to stop the search after hitting endVertex
///
__global__  void CUDA_SSSP_KERNEL2(  int *vertexArray, int *edgeArray, float *weightArray,
                                     unsigned char *maskArray, float *costArray, float *updatingCostArray)
{
    // access thread id
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (costArray[tid] > updatingCostArray[tid])
    {
        costArray[tid] = updatingCostArray[tid];
        maskArray[tid] = 1;
    }

    updatingCostArray[tid] = costArray[tid];
}

///////////////////////////////////////////////////////////////////////////////
//
//  Private Functions
//
//

///
/// Round Up Division function
///
size_t roundUp(int group_size, int global_size) 
{
    int r = global_size % group_size;
    if(r == 0) 
    {
        return global_size;
    } 
    else 
    {
        return global_size + group_size - r;
    }
}

///
/// Check whether the mask array is empty.  This tells the algorithm whether
/// it needs to continue running or not.
///
bool maskArrayEmpty(unsigned char *maskArray, int count)
{
    for(int i = 0; i < count; i++ )
    {
        if (maskArray[i] == 1)
        {
            return false;
        }
    }

    return true;
}

///
///  Allocate memory for input CUDA buffers and copy the data into device memory
///
void allocateCUDABuffers(GraphData *graph,
                         int **vertexArrayDevice, int **edgeArrayDevice, float **weightArrayDevice,
                         unsigned char **maskArrayDevice, float **costArrayDevice, float **updatingCostArrayDevice,
                         float **infinitiArrayDevice, int globalWorkSize)
{
    // V
    cutilSafeCall( hipMalloc( (void**) vertexArrayDevice, sizeof(int) * graph->vertexCount) );
    cutilSafeCall( hipMemcpy( *vertexArrayDevice, graph->vertexArray, sizeof(int) * graph->vertexCount, hipMemcpyHostToDevice) );

    // E
    cutilSafeCall( hipMalloc( (void**) edgeArrayDevice, sizeof(int) * graph->edgeCount) );
    cutilSafeCall( hipMemcpy( *edgeArrayDevice, graph->edgeArray, sizeof(int) * graph->edgeCount, hipMemcpyHostToDevice) );

    // W
    cutilSafeCall( hipMalloc( (void**) weightArrayDevice, sizeof(float) * graph->edgeCount) );
    cutilSafeCall( hipMemcpy( *weightArrayDevice, graph->weightArray, sizeof(float) * graph->edgeCount, hipMemcpyHostToDevice) );

    // M, C, U
    cutilSafeCall( hipMalloc( (void**) maskArrayDevice, sizeof(unsigned char) * globalWorkSize) );
    cutilSafeCall( hipMalloc( (void**) costArrayDevice, sizeof(float) * globalWorkSize) );
    cutilSafeCall( hipMalloc( (void**) updatingCostArrayDevice, sizeof(float) * globalWorkSize) );

    // This is quite annoying, but at the moment I can't find a way to set a float
    // value to a buffer in CUDA (cudaMemSet operates on bytes).  So I create an
    // infiniti array that is used to clear the buffers
    float *infinityArray = (float*) malloc(sizeof(float) * globalWorkSize);
    for(int i = 0; i < globalWorkSize; i++)
    {
        infinityArray[i] = FLT_MAX;
    }

    cutilSafeCall( hipMalloc( (void**) infinitiArrayDevice, sizeof(float) * globalWorkSize) );
    cutilSafeCall( hipMemcpy( *infinitiArrayDevice,infinityArray, sizeof(float) * globalWorkSize, hipMemcpyHostToDevice) );

    free (infinityArray);
}

///
/// Initialize CUDA buffers for single run of Dijkstra
///
void initializeCUDABuffers(GraphData *graph, int sourceVertex,
                           unsigned char *maskArrayDevice, float *costArrayDevice, float *updatingCostArrayDevice,
                           float *infinityArrayDevice, int globalWorkSize)
{
    hipMemset( maskArrayDevice, 0, sizeof(unsigned char) * graph->vertexCount );

    // FUTURE OPTIMIZATION: Figure out how to do this with a memset, or at least something not requiring a
    //                      full memcpy.
    hipMemcpy( costArrayDevice, infinityArrayDevice, sizeof(float) * globalWorkSize, hipMemcpyDeviceToDevice );
    hipMemcpy( updatingCostArrayDevice, infinityArrayDevice, sizeof(float) * globalWorkSize, hipMemcpyDeviceToDevice );

    // Set M[S] = true, C[S] = 0, U[S] = 0
    hipMemset( &maskArrayDevice[sourceVertex], 1, sizeof(unsigned char) );
    hipMemset( &costArrayDevice[sourceVertex], 0, sizeof(float) );
    hipMemset( &updatingCostArrayDevice[sourceVertex], 0, sizeof(float) );
}

///
/// Worker thread for running the algorithm on one of the GPUs
///
CUT_THREADPROC dijkstraThread(GPUPlan *plan)
{
    // Set GPU device
    cutilSafeCall( hipSetDevice(plan->device) );

    runDijkstra( plan->graph, plan->sourceVertices, 
                 plan->outResultCosts, plan->numResults );

}


///////////////////////////////////////////////////////////////////////////////
//
//  Public Functions
//
//

///
/// Run Dijkstra's shortest path on the GraphData provided to this function.  This
/// function will compute the shortest path distance from sourceVertices[n] ->
/// endVertices[n] and store the cost in outResultCosts[n].  The number of results
/// it will compute is given by numResults.
///
/// This function will run the algorithm on a single GPU.
///
/// \param graph Structure containing the vertex, edge, and weight arra
///              for the input graph
/// \param startVertices Indices into the vertex array from which to
///                      start the search
/// \param endVertices Indices into the vertex array from which to end
///                    the search.
/// \param outResultsCosts A pre-allocated array where the results for
///                        each shortest path search will be written
/// \param numResults Should be the size of all three passed inarrays
///
void runDijkstra( GraphData* graph, int *sourceVertices, float *outResultCosts, int numResults )
{
    int *vertexArrayDevice;
    int *edgeArrayDevice;
    float *weightArrayDevice;
    unsigned char *maskArrayDevice;
    float *costArrayDevice;
    float *updatingCostArrayDevice;
    float *infinityArrayDevice;



    // Set # of work items in work group and total in 1 dimensional range
    size_t localWorkSize = 512;
    size_t globalWorkSize = roundUp(localWorkSize, graph->vertexCount);
    
    // Allocate buffers in Device memory
    allocateCUDABuffers( graph, &vertexArrayDevice, &edgeArrayDevice, &weightArrayDevice,
                         &maskArrayDevice, &costArrayDevice, &updatingCostArrayDevice,
                         &infinityArrayDevice, globalWorkSize);

    unsigned char *maskArrayHost = (unsigned char*) malloc(sizeof(unsigned char) * graph->vertexCount);

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    for ( int i = 0 ; i < numResults; i++ )
    {
        // Initialize mask array to false, C and U to infiniti
        initializeCUDABuffers( graph, sourceVertices[i],
                              maskArrayDevice, costArrayDevice, updatingCostArrayDevice,
                              infinityArrayDevice, globalWorkSize);
        
        hipMemcpy( maskArrayHost, maskArrayDevice, sizeof(unsigned char) * graph->vertexCount, hipMemcpyDeviceToHost );

        while(!maskArrayEmpty(maskArrayHost, graph->vertexCount))
        {
            int gridSize = globalWorkSize / localWorkSize;

            dim3  threads( localWorkSize, 1, 1);
            dim3  grid( gridSize, 1, 1);    
            // execute the kernel
            CUDA_SSSP_KERNEL1<<< grid, threads >>>( vertexArrayDevice, edgeArrayDevice, weightArrayDevice,
                                                    maskArrayDevice, costArrayDevice, updatingCostArrayDevice,
                                                    graph->vertexCount, graph->edgeCount );
            CUT_CHECK_ERROR("CUDA_SSSP_KERNEL1");

            CUDA_SSSP_KERNEL2<<< grid, threads >>>( vertexArrayDevice, edgeArrayDevice, weightArrayDevice,
                                                    maskArrayDevice, costArrayDevice, updatingCostArrayDevice );
            CUT_CHECK_ERROR("CUDA_SSSP_KERNEL2");

            hipMemcpy( maskArrayHost, maskArrayDevice, sizeof(unsigned char) * graph->vertexCount, hipMemcpyDeviceToHost );
        }

        // Copy the result back
        cutilSafeCall( hipMemcpy( &outResultCosts[i * graph->vertexCount], &costArrayDevice[0], sizeof(float) * graph->vertexCount, hipMemcpyDeviceToHost) );
    }

    cutilCheckError(cutStopTimer(timer));
    //printf("Kernel GPU Processing time: %f (ms) \n", cutGetTimerValue(timer));

    free (maskArrayHost);

    // Free all the buffers
    cutilSafeCall(hipFree(vertexArrayDevice));
    cutilSafeCall(hipFree(edgeArrayDevice));
    cutilSafeCall(hipFree(weightArrayDevice));
    cutilSafeCall(hipFree(maskArrayDevice));
    cutilSafeCall(hipFree(costArrayDevice));
    cutilSafeCall(hipFree(updatingCostArrayDevice));
    cutilSafeCall(hipFree(infinityArrayDevice));
}



///
/// Run Dijkstra's shortest path on the GraphData provided to this function.  This
/// function will compute the shortest path distance from sourceVertices[n] ->
/// endVertices[n] and store the cost in outResultCosts[n].  The number of results
/// it will compute is given by numResults.
///
/// This function will run the algorithm on as many GPUs as is available.  It will
/// create N threads, one for each GPU, and chunk the workload up to perform
/// (numResults / N) searches per GPU.
///
/// \param graph Structure containing the vertex, edge, and weight arra
///              for the input graph
/// \param startVertices Indices into the vertex array from which to
///                      start the search
/// \param endVertices Indices into the vertex array from which to end
///                    the search.
/// \param outResultsCosts A pre-allocated array where the results for
///                        each shortest path search will be written
/// \param numResults Should be the size of all three passed inarrays
///
///

void runDijkstraMultiGPU( GraphData* graph, int *sourceVertices, 
                          float *outResultCosts, int numResults )
{
    int numGPUs;

    cutilSafeCall( hipGetDeviceCount(&numGPUs) );
    printf("CUDA-capable device count: %i\n", numGPUs);

    if (numGPUs == 0)
    {
        // ERORR: no GPUs present!
        return;
    }

    GPUPlan *gpuPlans = (GPUPlan*) malloc(sizeof(GPUPlan) * numGPUs);
    CUTThread *threadIDs = (CUTThread*) malloc(sizeof(CUTThread) * numGPUs);

    // Divide the workload out per GPU
    int resultsPerGPU = numResults / numGPUs;

    int offset = 0;

    for (int i = 0; i < numGPUs; i++)
    {
        gpuPlans[i].device = i;
        gpuPlans[i].graph = graph;
        gpuPlans[i].sourceVertices = &sourceVertices[offset];
        gpuPlans[i].outResultCosts = &outResultCosts[offset * graph->vertexCount];
        gpuPlans[i].numResults = resultsPerGPU;

        offset += resultsPerGPU;
    }

    // Add any remaining work to the last GPU
    if (offset < numResults)
    {
        gpuPlans[numGPUs - 1].numResults += (numResults - offset);
    }

    // Launch all the threads
    for (int i = 0; i < numGPUs; i++)
    {
        threadIDs[i] = cutStartThread((CUT_THREADROUTINE)dijkstraThread, (void*)(gpuPlans + i));
    }

    // Wait for the results from all threads
    cutWaitForThreads(threadIDs, numGPUs);

    free (gpuPlans);
    free (threadIDs);
}

///
/// Run Dijkstra's shortest path on the GraphData provided to this function.  This
/// function will compute the shortest path distance from sourceVertices[n] ->
/// endVertices[n] and store the cost in outResultCosts[n].  The number of results
/// it will compute is given by numResults.
///
/// This is a CPU *REFERENCE* implementation for use as a fallback.
///
/// \param graph Structure containing the vertex, edge, and weight arra
///              for the input graph
/// \param startVertices Indices into the vertex array from which to
///                      start the search
/// \param outResultsCosts A pre-allocated array where the results for
///                        each shortest path search will be written.
///                        This must be sized numResults * graph->numVertices.
/// \param numResults Should be the size of all three passed inarrays
///
void runDijkstraRef( GraphData* graph, int *sourceVertices,
                     float *outResultCosts, int numResults )
{

    // Create the arrays needed for processing the algorithm
    float *costArray = new float[graph->vertexCount];
    float *updatingCostArray = new float[graph->vertexCount];
    unsigned char *maskArray = new unsigned char[graph->vertexCount];

    for (int i = 0; i < numResults; i++)
    {
        // Initialize the buffer for this run
        for (int v = 0; v < graph->vertexCount; v++)
        {
            if (v == sourceVertices[i])
            {
                maskArray[v] = 1;
                costArray[v] = 0.0;
                updatingCostArray[v] = 0.0;
            }
            else
            {
                maskArray[v] = 0;
                costArray[v] = FLT_MAX;
                updatingCostArray[v] = FLT_MAX;
            }
        }

        while(!maskArrayEmpty(maskArray, graph->vertexCount))
        {
            // Equivalent of OCL_SSSP_KERNEL1()
            for (int tid = 0; tid < graph->vertexCount; tid++)
            {
                if ( maskArray[tid] != 0 )
                {
                    maskArray[tid] = 0;

                    int edgeStart = graph->vertexArray[tid];
                    int edgeEnd;
                    if (tid + 1 < (graph->vertexCount))
                    {
                        edgeEnd = graph->vertexArray[tid + 1];
                    }
                    else
                    {
                        edgeEnd = graph->edgeCount;
                    }

                    for(int edge = edgeStart; edge < edgeEnd; edge++)
                    {
                        int nid = graph->edgeArray[edge];

                        // One note here: whereas the paper specified weightArray[nid], I
                        //  found that the correct thing to do was weightArray[edge].  I think
                        //  this was a typo in the paper.  Either that, or I misunderstood
                        //  the data structure.
                        if (updatingCostArray[nid] > (costArray[tid] + graph->weightArray[edge]))
                        {
                            updatingCostArray[nid] = (costArray[tid] + graph->weightArray[edge]);
                        }
                    }
                }
            }

            // Equivalent of OCL_SSSP_KERNEL2()
            for (int tid = 0; tid < graph->vertexCount; tid++)
            {
                if (costArray[tid] > updatingCostArray[tid])
                {
                    costArray[tid] = updatingCostArray[tid];
                    maskArray[tid] = 1;
                }

                updatingCostArray[tid] = costArray[tid];
            }
        }

        // Copy the result back
        memcpy(&outResultCosts[i * graph->vertexCount], costArray, sizeof(float) * graph->vertexCount);
    }

    // Free temporary computation buffers
    delete [] costArray;
    delete [] updatingCostArray;
    delete [] maskArray;
}




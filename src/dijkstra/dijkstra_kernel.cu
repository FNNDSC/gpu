#include "hip/hip_runtime.h"
//
//
//  Description:
//      Implementation of Dijkstra's Single-Source Shortest Path (SSSP) algorithm on the GPU.
//      The basis of this implementation is the paper:
//
//          "Accelerating large graph algorithms on the GPU using CUDA" by
//          Parwan Harish and P.J. Narayanan
//
//
//  Author:
//      Dan Ginsburg
//
//  Children's Hospital Boston
//  GPL v2
//
#ifndef DIJKSTRA_KERNEL_H
#define DIJKSTRA_KERNEL_H

#include <stdio.h>
#include <multithreading.h>

#define INFINITI    (9999999) // Should really fix how infiniti is set for the float buffers.  Will do this in the
                              // real version.

///
//  Types
//
//
//  This data structure and algorithm implementation is based on
//  Accelerating large graph algorithms on the GPU using CUDA by
//  Parwan Harish and P.J. Narayanan
//
typedef struct
{
    // (V) This contains a pointer to the edge list for each vertex
    int *vertexArray;

    // Vertex count
    int vertexCount;

    // (E) This contains pointers to the vertices that each edge is attached to
    int *edgeArray;

    // Edge count
    int edgeCount;

    // (W) Weight array
    float *weightArray;

} GraphData;

// This structure is used in the multi-GPU implementation of the algorithm.
// This structure defines the workload for each GPU.  The code chunks up
// the work on a per-GPU basis.
typedef struct
{
    // GPU number to run algorithm on
    int device;

    // Pointer to graph data
    GraphData *graph;

    // Source vertex indices to process
    int *sourceVertices;

    // End vertex indices to process
    int *endVertices;

    // Results of processing
    float *outResultCosts;

    // Number of results
    int numResults;

} GPUPlan;

///
/// This is part 1 of the Kernel from Algorithm 4 in the paper
///
__global__  void CUDA_SSSP_KERNEL1( int *vertexArray, int *edgeArray, float *weightArray,
                                    unsigned char *maskArray, float *costArray, float *updatingCostArray,
                                    int vertexCount, int edgeCount )
{
    // access thread id
    unsigned int tid = threadIdx.x;

    if ( maskArray[tid] != 0 )
    {
        maskArray[tid] = 0;

        int edgeStart = vertexArray[tid];
        int edgeEnd;
        if (tid + 1 < (vertexCount))
        {
            edgeEnd = vertexArray[tid + 1];
        }
        else
        {
            edgeEnd = edgeCount;
        }

        for(int edge = edgeStart; edge < edgeEnd; edge++)
        {
            int nid = edgeArray[edge];

            // One note here: whereas the paper specified weightArray[nid], I
            //  found that the correct thing to do was weightArray[edge].  I think
            //  this was a typo in the paper.  Either that, or I misunderstood
            //  the data structure.
            if (updatingCostArray[nid] > (costArray[tid] + weightArray[edge]))
            {
                updatingCostArray[nid] = (costArray[tid] + weightArray[edge]);
            }
        }
    }
}

///
/// This is part 2 of the Kernel from Algorithm 5 in the paper
///
__global__  void CUDA_SSSP_KERNEL2(  int *vertexArray, int *edgeArray, float *weightArray,
                                     unsigned char *maskArray, float *costArray, float *updatingCostArray,
                                     int endVertex )
{
    // access thread id
    unsigned int tid = threadIdx.x;

    if (costArray[tid] > updatingCostArray[tid])
    {
        costArray[tid] = updatingCostArray[tid];

        // Stop if we have hit the final vertex
        if (tid != endVertex)
        {
            maskArray[tid] = 1;
        }
    }

    updatingCostArray[tid] = costArray[tid];
}

///
/// Check whether the mask array is empty.  This tells the algorithm whether
/// it needs to continue running or not.
///
bool maskArrayEmpty(unsigned char *maskArray, int count)
{
    for(int i = 0; i < count; i++ )
    {
        if (maskArray[i] == 1)
        {
            return false;
        }
    }

    return true;
}

///
///  Allocate memory for input CUDA buffers and copy the data into device memory
///
void allocateCUDABuffers(GraphData *graph,
                         int **vertexArrayDevice, int **edgeArrayDevice, float **weightArrayDevice,
                         unsigned char **maskArrayDevice, float **costArrayDevice, float **updatingCostArrayDevice)
{
    // V
    cutilSafeCall( hipMalloc( (void**) vertexArrayDevice, sizeof(int) * graph->vertexCount) );
    cutilSafeCall( hipMemcpy( *vertexArrayDevice, graph->vertexArray, sizeof(int) * graph->vertexCount, hipMemcpyHostToDevice) );

    // E
    cutilSafeCall( hipMalloc( (void**) edgeArrayDevice, sizeof(int) * graph->edgeCount) );
    cutilSafeCall( hipMemcpy( *edgeArrayDevice, graph->edgeArray, sizeof(int) * graph->edgeCount, hipMemcpyHostToDevice) );

    // W
    cutilSafeCall( hipMalloc( (void**) weightArrayDevice, sizeof(float) * graph->edgeCount) );
    cutilSafeCall( hipMemcpy( *weightArrayDevice, graph->weightArray, sizeof(float) * graph->edgeCount, hipMemcpyHostToDevice) );

    // M, C, U
    cutilSafeCall( hipMalloc( (void**) maskArrayDevice, sizeof(unsigned char) * graph->vertexCount) );
    cutilSafeCall( hipMalloc( (void**) costArrayDevice, sizeof(float) * graph->vertexCount) );
    cutilSafeCall( hipMalloc( (void**) updatingCostArrayDevice, sizeof(float) * graph->vertexCount) );
}

///
/// Initialize CUDA buffers for single run of Dijkstra
///
void initializeCUDABuffers(GraphData *graph, int sourceVertex,
                           unsigned char *maskArrayDevice, float *costArrayDevice, float *updatingCostArrayDevice)
{
    hipMemset( maskArrayDevice, 0, sizeof(unsigned char) * graph->vertexCount );
    hipMemset( costArrayDevice, INFINITI, sizeof(float) * graph->vertexCount ); // This needs to be replaced, not correct for float
    hipMemset( updatingCostArrayDevice, INFINITI, sizeof(float) * graph->vertexCount ); // This needs to be replaced, not correct for float

    // Set M[S] = true, C[S] = 0, U[S] = 0
    hipMemset( &maskArrayDevice[sourceVertex], 1, sizeof(unsigned char) );
    hipMemset( &costArrayDevice[sourceVertex], 0, sizeof(float) );
    hipMemset( &updatingCostArrayDevice[sourceVertex], 0, sizeof(float) );
}

///
/// Run Dijkstra's shortest path on the GraphData provided to this function.  This function
/// assumes that the caller has allocated GPU memory for each of the arrays and has placed
/// vertex, edge, and weight data in each of the arrays.  This will determine the shortest
/// path from the sourceVertex to any other vertices.
///
/// This is a prototype that will be developed further when I optimize mris_pmake.
///
void runDijkstra( GraphData* graph, int *sourceVertices, int *endVertices,
                   float *outResultCosts, int numResults)
{
    int *vertexArrayDevice;
    int *edgeArrayDevice;
    float *weightArrayDevice;
    unsigned char *maskArrayDevice;
    float *costArrayDevice;
    float *updatingCostArrayDevice;


    // Allocate buffers in Device memory
    allocateCUDABuffers( graph, &vertexArrayDevice, &edgeArrayDevice, &weightArrayDevice,
                         &maskArrayDevice, &costArrayDevice, &updatingCostArrayDevice );

    unsigned char *maskArrayHost = (unsigned char*) malloc(sizeof(unsigned char) * graph->vertexCount);

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    for ( int i = 0 ; i < numResults; i++ )
    {
        // Initialize mask array to false, C and U to infiniti
        initializeCUDABuffers( graph, sourceVertices[i],
                              maskArrayDevice, costArrayDevice, updatingCostArrayDevice );


        dim3  grid( 1, 1, 1);
        dim3  threads( graph->vertexCount, 1, 1);


        hipMemcpy( maskArrayHost, maskArrayDevice, sizeof(unsigned char) * graph->vertexCount, hipMemcpyDeviceToHost );

        while(!maskArrayEmpty(maskArrayHost, graph->vertexCount))
        {
            // execute the kernel
            CUDA_SSSP_KERNEL1<<< grid, threads >>>( vertexArrayDevice, edgeArrayDevice, weightArrayDevice,
                                                    maskArrayDevice, costArrayDevice, updatingCostArrayDevice,
                                                    graph->vertexCount, graph->edgeCount );
            CUT_CHECK_ERROR("CUDA_SSSP_KERNEL1");

            CUDA_SSSP_KERNEL2<<< grid, threads >>>( vertexArrayDevice, edgeArrayDevice, weightArrayDevice,
                                                    maskArrayDevice, costArrayDevice, updatingCostArrayDevice,
                                                    endVertices[i] );
            CUT_CHECK_ERROR("CUDA_SSSP_KERNEL2");

            hipMemcpy( maskArrayHost, maskArrayDevice, sizeof(unsigned char) * graph->vertexCount, hipMemcpyDeviceToHost );
        }

        float result;

        // Copy the result back
        cutilSafeCall( hipMemcpy( &result, &costArrayDevice[endVertices[i]], sizeof(float), hipMemcpyDeviceToHost) );
        outResultCosts[i] = result;
    }

    cutilCheckError(cutStopTimer(timer));
    printf("Kernel GPU Processing time: %f (ms) \n", cutGetTimerValue(timer));

    free (maskArrayHost);

    // Free all the buffers
    cutilSafeCall(hipFree(vertexArrayDevice));
    cutilSafeCall(hipFree(edgeArrayDevice));
    cutilSafeCall(hipFree(weightArrayDevice));
    cutilSafeCall(hipFree(maskArrayDevice));
    cutilSafeCall(hipFree(costArrayDevice));
    cutilSafeCall(hipFree(updatingCostArrayDevice));
}

///
/// Worker thread for running the algorithm on one of the GPUs
///
CUT_THREADPROC dijkstraThread(GPUPlan *plan)
{
    // Set GPU device
    cutilSafeCall( hipSetDevice(plan->device) );

    runDijkstra( plan->graph, plan->sourceVertices, plan->endVertices,
                 plan->outResultCosts, plan->numResults );

}

///
/// Multi-GPU version of Dijkstra's algorithm that takes a list of source/end vertices
/// and produces a
///
void runDijkstraMultiGPU( GraphData* graph, int *sourceVertices, int *endVertices,
                          float *outResultCosts, int numResults )
{
    int numGPUs;

    cutilSafeCall( hipGetDeviceCount(&numGPUs) );
    printf("CUDA-capable device count: %i\n", numGPUs);

    if (numGPUs == 0)
    {
        // ERORR: no GPUs present!
        return;
    }

    GPUPlan *gpuPlans = (GPUPlan*) malloc(sizeof(GPUPlan) * numGPUs);
    CUTThread *threadIDs = (CUTThread*) malloc(sizeof(CUTThread) * numGPUs);

    // Divide the workload out per GPU
    int resultsPerGPU = numResults / numGPUs;

    int offset = 0;

    for (int i = 0; i < numGPUs; i++)
    {
        gpuPlans[i].device = i;
        gpuPlans[i].graph = graph;
        gpuPlans[i].sourceVertices = &sourceVertices[offset];
        gpuPlans[i].endVertices = &endVertices[offset];
        gpuPlans[i].outResultCosts = &outResultCosts[offset];
        gpuPlans[i].numResults = resultsPerGPU;

        offset += resultsPerGPU;
    }

    // Add any remaining work to the last GPU
    if (offset < numResults)
    {
        gpuPlans[numGPUs - 1].numResults += (numResults - offset);
    }

    // Launch all the threads
    for (int i = 0; i < numGPUs; i++)
    {
        threadIDs[i] = cutStartThread((CUT_THREADROUTINE)dijkstraThread, (void*)(gpuPlans + i));
    }

    // Wait for the results from all threads
    cutWaitForThreads(threadIDs, numGPUs);

    free (gpuPlans);
    free (threadIDs);
}

#endif // #ifndef DIJKSTRA_KERNEL_H
